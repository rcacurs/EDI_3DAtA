#include "hip/hip_runtime.h"
#include "../include/imageMatrix.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
//ImMatG function definition

ImMatG::ImMatG(){
	rows = 0;
	cols = 0;
}
ImMatG::ImMatG(size_t rows, size_t cols, double * data, bool onDeviceMemory){
	this->rows = rows;
	this->cols = cols;
	if (onDeviceMemory){
		this->data_d = data;
	} else{
		hipMalloc(&(this->data_d), rows*cols*sizeof(double));
		hipError_t cuerror = hipMemcpy(this->data_d, data, rows*cols*sizeof(double), hipMemcpyHostToDevice);
	}
}
ImMatG::ImMatG(size_t rows, size_t cols){
	this->rows = rows;
	this->cols = cols;
	hipMalloc(&(this->data_d), rows*cols*sizeof(double));
}
ImMatG::~ImMatG(){
	hipFree((this->data_d));
}
size_t ImMatG::getLength(void){
	return rows*cols;
}

// GPU KERNELS
__global__ void transposeKernel(const double *input, double *output, int height, int width){

	extern __shared__ double temp[];
	int xIndex = blockIdx.x*blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y*blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height)){
		int id_in = yIndex*width + xIndex;
		temp[threadIdx.x+threadIdx.y*(blockDim.x)] = input[id_in];
	}

	__syncthreads();

	int tempXIndex = xIndex;
	xIndex = yIndex;
	yIndex = tempXIndex;

	if ((xIndex < height) && (yIndex < width)){
		int id_out = xIndex+yIndex*height;
		output[id_out] = temp[threadIdx.x+threadIdx.y*(blockDim.x)];
	}
}

ImMatG* ImMatG::transpose(){
	ImMatG *result= new ImMatG(cols, rows);
	int numThreads = 16;
	int blocksX = ceil(((float)cols) / numThreads);
	int blocksY = ceil(((float)rows) / numThreads);

	transposeKernel<<<dim3(blocksX, blocksY, 1), dim3(numThreads, numThreads, 1), (numThreads)*(numThreads)*sizeof(double)>>>(data_d, result->data_d, rows, cols);
	return result;
}

__global__ void fillRowKernel(double *data, size_t cols, size_t row, double value){
	int Xidx = threadIdx.x + blockIdx.x*blockDim.x;
	if (Xidx < cols){
		data[Xidx + row*cols] = value;
	}
}

void ImMatG::fillRow(size_t row, double value){
	if ((row >= this->rows) || (row < 0)){
		std::cout << "Index doesn't agree with image size" << std::endl;
		return;
	}

	int threadNum = 128;
	fillRowKernel << <dim3(ceil(cols / threadNum), 1, 1), dim3(threadNum, 1, 1) >> >(data_d, cols, row, value);
}

__global__ void getColumnKernel(double *image, size_t rows, size_t cols, double *column){

	int xIdx = threadIdx.x + blockIdx.x*blockDim.x;
	int yIdx = threadIdx.y + blockIdx.y*blockDim.y;


	if ((xIdx > cols) && (yIdx < cols)){

	}
}

